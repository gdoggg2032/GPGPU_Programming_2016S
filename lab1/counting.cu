#include "hip/hip_runtime.h"
#include <stdio.h>
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/generate.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/host_vector.h>

//const int K = 500;
__device__ const int D = 9; // log(500)
const int L = 40000500;
struct is_one
{
	__host__ __device__
		bool operator()(const int x)
		{
			return (x == 1);
		}
};

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__device__ int BIT[40000500][D];


__global__ void BITBuilding1(const char *text, int *pos, int text_size)
{
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) % text_size;
	if(idx >= text_size)
		return;
	for(int d = 0; d < D; d++)
	{
		if(d==0)
		{

			if(text[idx] != '\n')
				BIT[idx][0] = 1;
			else
				BIT[idx][0] = 0;
		}
		else
		{
			
			int dim = (int)pow(2, d);
			if(idx < text_size/dim)
			{
				BIT[idx][d] = (BIT[2*idx][d-1] && BIT[(2*idx+1)][d-1]);
	
			}
		}

		__syncthreads();
	}
}
__global__ void BITBuilding(const char *text, int *pos, int text_size)
{
	//printf("BITBuilding\n");	
	int idx = (blockIdx.x * blockDim.x + threadIdx.x);
	if(idx >= text_size)
		return;
	//printf("building: index: %d\n", blockIdx.x * blockDim.x + threadIdx.x);
	/* test */
	
	/*88888 */
	//if(idx >= text_size)return;
	//int dim = 1;
	for(int d = 0; d < D; d++)
	{
		if(d == 0)
		{
			if(text[idx] != '\n')
				BIT[idx][d] = 1;
			else
				BIT[idx][d] = 0;

		}
		else
		{
			int dim = (int)pow(2, d);
			if(idx < text_size/dim)
			{
				BIT[idx][d] = (BIT[2*idx][d-1] && BIT[(2*idx+1)][d-1]);
			}

		}
		//dim = dim << 1;
		__syncthreads();
	}
}

__global__ void Counting(const char *text, int *pos, int text_size)
{
	//printf("Counting\n");
	int idx = (blockIdx.x * blockDim.x + threadIdx.x) % text_size;
	if(idx >= text_size)
		return;
	//printf("counting: index: %d\n", blockIdx.x * blockDim.x + threadIdx.x);
	//if(idx>=text_size)return;
	/* gpu part */
	if(BIT[idx][0] == 0)
		pos[idx] = 0;//*(pos+idx) = 0;
	else
	{
		int len = 0;

		int d = 0;
		int add = 1;
		int index = idx;


		while(1)
		{


			if(index <= 0)
				break;
			if(index%2 == 0)
			{
				index -= 1;
				len += add;
			}

			if(BIT[(index-1)/2][ d+1] == 1)
			{


				add *= 2;
				d += 1;
				index = (index-1)/2;
			}
			else
			{
				break;

			}

		}

		while(index >= 0 && add > 0 && d >= 0)
		{
			if(BIT[index][ d] == 1)
			{
				// to left-down
				len += add;
				index = index * 2 - 1;
				d -= 1;

			}
			else
			{
				// to right-down
				index = index * 2 + 1;
				d -= 1;

			}

			add /= 2;
		}

		pos[idx] = len;//*(pos+idx) = len;
	}

	//	__syncthreads();
}
__global__ void printTable(int text_size)
{
	printf("in printTable()\n");

	printf("%d \n", BIT[0][0]);
	for(int i = 0; i < 10; i++ ){
		for(int j = 0; j < D; j++)
			printf("%d ", BIT[i][j]);
		printf("\n");
	}
}
void CountPosition(const char *text, int *pos, int text_size)
{
	int threadNum = text_size;
	//BITBuilding<<<1, threadN>>>(text, pos, text_size );
	BITBuilding1<<<39063, 1024>>>(text, pos, text_size);
	printf("build\n");
	//BITBuilding1<<<10, 10>>>(text, pos, text_size);
	hipDeviceSynchronize();
	printf("print\n");
	printTable<<<1, 1>>>(text_size);
	hipDeviceSynchronize();
	printf("count\n");
	
	Counting<<<39063, 1024>>>(text, pos, text_size );
	//Counting<<<(text_size)/threadNum + 1, threadNum>>>(text, pos, text_size );
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	int nhead;
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head);
	
	thrust::device_vector<int> index_d(text_size);
	thrust::sequence(index_d.begin(), index_d.end());

	thrust::device_ptr<int> ret = thrust::copy_if(index_d.begin(), index_d.end(), pos_d, head_d, is_one());

	nhead = ret - head_d;
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{
}
